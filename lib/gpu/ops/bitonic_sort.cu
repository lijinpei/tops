#include "hip/hip_runtime.h"
#include "tops/gpu/error_check.hpp"
#include "tops/gpu/launch_grid.hpp"
#include "tops/gpu/ops/bitonic_sort.hpp"
#include "tops/gpu/ops/bitonic_sort_impl.cuh"
#include "tops/utils/int_numeric.hpp"

#include "ATen/ScalarType.h"
#include "ATen/Tensor.h"
#include "ATen/cuda/HIPContext.h"

#include "hip/hip_runtime.h"

#include <cassert>
#include <cstdint>
#include <utility>

namespace tops {
namespace gpu {

namespace {
struct BitonicSortConfig {
  static constexpr uint8_t VecCount = 4;
  static constexpr uint8_t NumelPerThread = 8;
  static constexpr uint8_t LanesPerRow = 32;
  static constexpr uint8_t WarpsPerRow = 32;
  static constexpr uint8_t WarpsOnCol = 1;
};
LaunchGrid decide_launch_grid(size_t dim0, size_t dim1, uint32_t num_sm) {
  uint32_t num_warps;
  if (dim1 >= 32 * 128) {
    num_warps = 32;
  } else {
    num_warps = next_power_of_2(dim1 / 128);
  }
  uint32_t boost_factor = 32 / num_warps;
  uint32_t num_blocks = std::min<size_t>(boost_factor * num_sm, dim0);
  return {num_blocks, num_warps};
}
} // namespace

void bitonic_sort(const at::Tensor &self, std::optional<bool> stable,
                  int64_t dim, bool descending, const at::Tensor &values,
                  const at::Tensor &indices) {
  assert(self.dtype() == c10::kFloat);
  assert(values.dtype() == c10::kFloat);
  assert(indices.dtype() == c10::kLong);

  auto num_dim = self.dim();
  if (dim < 0) {
    dim += num_dim;
  }
  assert(dim + 1 == num_dim);
  assert(!stable.has_value() || !stable.value());

  auto self_cont = self.contiguous();
  auto numel = self.numel();
  auto row_size = self.size(dim);
  auto col_size = numel / row_size;

  auto *dev_prop = at::cuda::getCurrentDeviceProperties();
  auto num_sm = dev_prop->multiProcessorCount;
  auto launch_grid = decide_launch_grid(col_size, row_size, num_sm);
  kernels::bitonic_sort<BitonicSortConfig>
      <<<launch_grid.grid_shape, launch_grid.block_shape>>>(
          values.data_ptr<float>(), indices.data_ptr<int64_t>(),
          self.data_ptr<float>(), col_size, row_size);
  ;
  CHECK_GPU_RUNTIME_API_CALL(hipGetLastError);
}

} // namespace gpu
} // namespace tops
