#include "hip/hip_runtime.h"
#include "tops/gpu/error_check.hpp"
#include "tops/gpu/special_reg.cuh"

#include "absl/flags/flag.h"
#include "absl/flags/parse.h"
#include "spdlog/spdlog.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "gtest/gtest.h"

#include <algorithm>
#include <cassert>
#include <stdexcept>

ABSL_FLAG(unsigned int, device, 0, "GPU device to run test on");

namespace {

hipDevice_t dev;

void init_gpu_device() {
  int dev_id = absl::GetFlag(FLAGS_device);
  CHECK_GPU_DRIVER_API_CALL(hipInit, 0);
  int dev_count;
  CHECK_GPU_DRIVER_API_CALL(hipGetDeviceCount, &dev_count);
  if (dev_id >= dev_count) {
    spdlog::error("request non exist device {} (only exists {} devices)",
                  dev_id, dev_count);
    throw std::runtime_error("invalid --device command line argument");
  }
  CHECK_GPU_DRIVER_API_CALL(hipDeviceGet, &dev, dev_id);
  CHECK_GPU_RUNTIME_API_CALL(hipSetDevice, dev_id);
}

void finish_gpu_device() { CHECK_GPU_RUNTIME_API_CALL(hipDeviceReset); }

__global__ void get_sm_id_kernel(int *result) {
  auto cta_id = tops::gpu::get_cta_x_id();
  int sm_id = tops::gpu::get_curr_phys_sm_id();
  result[cta_id] = sm_id;
}

} // namespace

TEST(SpecialReg, SM_ID) {
  int max_smem, sm_count;
  CHECK_GPU_DRIVER_API_CALL(
      hipDeviceGetAttribute, &max_smem,
      hipDeviceAttributeSharedMemPerBlockOptin, dev);
  CHECK_GPU_DRIVER_API_CALL(hipDeviceGetAttribute, &sm_count,
                            hipDeviceAttributeMultiprocessorCount, dev);
  CHECK_GPU_RUNTIME_API_CALL(hipFuncSetAttribute, &get_sm_id_kernel,
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             max_smem);
  thrust::device_vector<int> sm_ids(sm_count);
  get_sm_id_kernel<<<sm_count, 1, max_smem>>>(sm_ids.data().get());
  CHECK_GPU_DRIVER_API_CALL(hipStreamSynchronize, nullptr);
  thrust::host_vector<int> host_sm_ids = sm_ids;
  EXPECT_EQ(host_sm_ids.size(), sm_count);
  std::sort(host_sm_ids.begin(), host_sm_ids.end());
  for (int i = 0; i < sm_count; ++i) {
    EXPECT_EQ(i, host_sm_ids[i]);
  }
}

int main(int argc, char **argv) {
  testing::InitGoogleTest(&argc, argv);
  absl::ParseCommandLine(argc, argv);
  init_gpu_device();
  auto res = RUN_ALL_TESTS();
  finish_gpu_device();
  return res;
}
